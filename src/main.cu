#include "hip/hip_runtime.h"
#include "json.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <cstring>
#include <utility>
#include <tuple>
#include <cmath>
#include <chrono>
#include "config.h"
#include "io.h"


#ifdef _CUGRAPH
//#include <algorithms.hpp>
//#include <graph.hpp>
#endif

#include "edge_metrics.cu"
#include "edge_metrics_binning.h"
#include "utils.cuh"

#define NUM_THREADS  256
#define NUM_BLOCKS  1024

using namespace std;

typedef unsigned int READ_TYPE; // the format in which binary files are written
typedef unsigned int vid_t; // used for adj and xadj and SHOULD REPRESENT |E| WITHOUT OVERFLOWING
typedef float jac_t; // used for jaccards (or floats in general)
typedef unsigned long long ull;

int main(int argc, char** argv) {
#ifndef SORT_ASC
    cout << "Not sorting CSR by degrees\n";
#elif SORT_ASC==1
    cout << "Sorting CSR by degrees in ascending order\n";
#elif SORT_ASC==0
  cout << "Sorting CSR by degrees in descending order\n";
#else
  cout << "Passed an illegal value for sorting of CSR - undefining the sort variable\n";
#undef SORT_ASC
#endif
#if (!_DIRECTED)
    cout << "Treating graph as an undirected graph\n";
#elif (_DIRECTED)
    cout << "Treating graph as a directed graph\n";
#else
  cout << "Bad directed preprocessor value. Exiting\n";
  return 1;
#endif
  string output_json_file_name, input_graph_file_name;
  #ifdef _BINNING
  string binning_experiment_json_file_name;
  #endif
  int num_average;
  if (!parse_arguments(argc, const_cast<const char**>(argv), input_graph_file_name,
  #ifdef _BINNING
  binning_experiment_json_file_name,
  #endif 
  output_json_file_name, num_average)){
    return 1;
  }
  cout << "Using the average of " << num_average << " runs" << endl;

  // Prepare output json
  JSONWrapper output_json;
  // If it's already been written to, read it back, otherwise initialize it
  if (!check_file_exists(output_json_file_name)){
      output_json = initialize_output_json(input_graph_file_name);
  } else {
      output_json = read_json(output_json_file_name);
  }
  cout << "Printing results to the file " << output_json_file_name;

  // Reading graph
  cout << endl << endl << "Graph: " << input_graph_file_name << endl;
  graph<vid_t, vid_t> g = open_graph<vid_t, READ_TYPE>(input_graph_file_name, _DIRECTED);
  print_graph_statistics<vid_t, READ_TYPE>(g, output_json);
  cout << "##############################" << endl << endl;

  pretty_print_results(cout, "Algorithm", "Time", "Errors");


  // Create Jaccard array -- will contain ground truth
  jac_t* emetrics = new jac_t[g.m];

  // Check if the jaccard value has been cached before
  string jaccards_output_path = string(input_graph_file_name)+ ".corr.bin";
  ifstream infile_corr_bin(jaccards_output_path , ios::in | ios::binary);

  bool have_correct = false;
  if(infile_corr_bin.is_open()) {
      cout << "Reading correct jaccard values from disk\n";
      have_correct = true;
      infile_corr_bin.read((char*)(emetrics), sizeof(jac_t)*g.m);
  }
  double total_time = 0;
  double start, end;
#ifdef _CPU
    jac_t* emetrics_vanilla = new jac_t[g.m];
  for (int i = 0; i< num_average; i++){
    start = omp_get_wtime();
    edge_based_metrics<_DIRECTED, vid_t, vid_t, jac_t>(g.is, g.xadj, g.adj, g.n, emetrics_vanilla);
    end = omp_get_wtime();
    total_time+=end-start;
  }
  validate_and_write(g, "CPU", emetrics, emetrics_vanilla, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);

  //Compute edge-based metrics
  jac_t* emetrics_bitmap = new jac_t[g.m];
  total_time = 0;
  for (int i = 0; i< num_average; i++){
    start = omp_get_wtime();
    edge_based_metrics_bitmap<_DIRECTED, vid_t, vid_t, jac_t>(g.is, g.xadj, g.adj, g.n, emetrics_bitmap);
    end = omp_get_wtime();
    total_time+=end-start;
  }
  validate_and_write(g, "CPU - bitmap", emetrics, emetrics_bitmap, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);

#endif
#ifdef _GPU
    // prepare GPU
  int device_id = get_device_id(0);
  hipSetDevice(device_id);
  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device_id);
  const int max_sm = props.sharedMemPerBlock;
  cout << device_id << ": " << props.name << ": " << props.major << "." << props.minor << " - Shared memory per block (B): " << max_sm << endl;
  // Lazy starting the GPU
  float *dummy;
  gpuErrchk( hipMalloc((void**)&dummy, sizeof(float) ) );
  //preparation for cuda
  graph<vid_t, vid_t> g_d;
  double alloc_copy_start = omp_get_wtime();
  cout << "GPU: allocating xadj\n";
  gpuErrchk( vcudaMalloc((void**)&g_d.xadj, sizeof(vid_t) * (g.n + 1) ) );
  gpuErrchk( hipMemcpy(g_d.xadj, g.xadj, sizeof(vid_t) * (g.n + 1), hipMemcpyHostToDevice) );
  //cout << "GPU: allocating tadj\n";
  //gpuErrchk( vcudaMalloc((void**)&g_d.tadj, sizeof(vid_t) * (g.n + 1) ) );
  //gpuErrchk( hipMemcpy(g_d.tadj, tadj, sizeof(vid_t) * (g.n + 1), hipMemcpyHostToDevice) );
  cout << "GPU: allocating adj\n";
  gpuErrchk( vcudaMalloc((void**)&g_d.adj, sizeof(vid_t) * g.m ) );
  gpuErrchk( hipMemcpy(g_d.adj, g.adj, sizeof(vid_t) * g.m, hipMemcpyHostToDevice) );
  jac_t* emetrics_cuda = new jac_t[(ull)g.m], *emetrics_cuda_d, *emetrics_cuda_r, *emetrics_cuda_k;
  cout << "GPU: allocating emetrics\n";
  gpuErrchk( vcudaMalloc((void**)&emetrics_cuda_d, sizeof(jac_t) * (ull)(g.m)) );
  gpuErrchk( vcudaMalloc((void**)&emetrics_cuda_r, sizeof(jac_t) * (ull)(g.m*7)) );
  gpuErrchk( vcudaMalloc((void**)&emetrics_cuda_k, sizeof(jac_t) * (ull)(g.m*3)) );

  double alloc_copy_end = omp_get_wtime();
  double t = alloc_copy_end -alloc_copy_start; 
  output_json.SetJSONNested("experiments", "GPU - alloc/copy", get_result_json(t, 0));
  write_json_to_file(output_json_file_name, output_json);
  pretty_print_results(cout, "GPU - alloc/copy" , to_string(t), to_string(0));

  alloc_copy_start = omp_get_wtime();
  cout << "GPU: allocating is\n";
  gpuErrchk( vcudaMalloc((void**)&g_d.is, sizeof(vid_t) * g.m ) );
  gpuErrchk( hipMemcpy(g_d.is, g.is, sizeof(vid_t) * g.m, hipMemcpyHostToDevice) );
  alloc_copy_end = omp_get_wtime();
  t = alloc_copy_end -alloc_copy_start; 
  pretty_print_results(cout, "GPU - alloc/copy is" , to_string(t), to_string(0));
  output_json.SetJSONNested("experiments", "GPU - alloc/copy", get_result_json(t, 0));
  write_json_to_file(output_json_file_name, output_json);
  if (_DIRECTED == 0){
    alloc_copy_start = omp_get_wtime();
    cout << "GPU: allocating xadj_start\n";
    gpuErrchk( vcudaMalloc((void**)&g_d.xadj_start, sizeof(vid_t) * (g.n) ) );
    gpuErrchk( hipMemcpy(g_d.xadj_start, g.xadj_start, sizeof(vid_t) * (g.n), hipMemcpyHostToDevice) );
    alloc_copy_end = omp_get_wtime();
    t = alloc_copy_end -alloc_copy_start; 
    pretty_print_results(cout, "GPU - alloc/copy xadj_start" , to_string(t), to_string(0));
      output_json.SetJSONNested("experiments", "GPU - alloc/copy xadj_start", get_result_json(t, 0));
      write_json_to_file(output_json_file_name, output_json);
  }

#ifdef _SIMPLE_GPU_EDGE
  gpuErrchk( hipMemset(emetrics_cuda_d, 0, sizeof(jac_t) * g.m) );
  total_time = 0;
  int gg = 32;
  int a = 1;
  {
    dim3 grid(1,1,1);
    grid.y = min(MAX_GRID_DIM, g.m/a);
    grid.z = min(MAX_GRID_DIM, max(1, g.m/a/grid.y));
    for (int i = 0; i< num_average; i++){
      start = omp_get_wtime();
      jac_edge_based_small<_DIRECTED, vid_t, vid_t, jac_t><<<grid, gg>>>(g_d.xadj, g_d.adj, g_d.is, g.n, emetrics_cuda_d);
      gpuErrchk( hipDeviceSynchronize() );
      gpuErrchk( hipMemcpy(emetrics_cuda, emetrics_cuda_d, (ull)sizeof(jac_t) * g.m, hipMemcpyDeviceToHost) );
      end = omp_get_wtime();
      total_time+=end-start;
    }
  }
  validate_and_write(g, "GPU - SG per edge g="+to_string(gg)+" a="+to_string(a), emetrics, emetrics_cuda, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);

#endif
 
#ifdef _SIMPLE_GPU
  //Compute edge-based metrics cuda
  gpuErrchk( hipMemset(emetrics_cuda_r, 0, (sizeof(jac_t) * g.m * 7)) );
  total_time = 0;
  for (int i = 0; i< num_average; i++){
    start = omp_get_wtime();
    edge_based_on_device<_DIRECTED, vid_t, vid_t, jac_t><<<NUM_BLOCKS, NUM_THREADS>>>(g_d.is, g_d.xadj, g_d.adj, g.n, emetrics_cuda_d, 1);
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpy(emetrics_cuda_r, emetrics_cuda_d, (ull)sizeof(jac_t) * g.m * 7, hipMemcpyDeviceToHost) );
    end = omp_get_wtime();
    total_time+=end-start;
  }
// if no CPU runs are happening, set the reference jaccard values (for error checking) to be this kernel's
  validate_and_write(g,  "GPU - Compute on Device", emetrics, emetrics_cuda, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);
  

  gpuErrchk( hipMemset(emetrics_cuda_k, 0, (sizeof(jac_t) * g.m*3)) );
  total_time = 0;
  for (int i = 0; i< num_average; i++){
    start = omp_get_wtime();
    edge_based_on_device<_DIRECTED, vid_t, vid_t, jac_t><<<NUM_BLOCKS, NUM_THREADS>>>(g_d.is, g_d.xadj, g_d.adj, g.n, emetrics_cuda_d, 1);
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpy(emetrics_cuda_k, emetrics_cuda_d, (ull)sizeof(jac_t) * g.m * 3, hipMemcpyDeviceToHost) );
    end = omp_get_wtime();
    total_time+=end-start;
  }
// if no CPU runs are happening, set the reference jaccard values (for error checking) to be this kernel's
  validate_and_write(g,  "GPU - Compute on Host", emetrics, emetrics_cuda, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);
#endif

#ifdef _DONGARRA
  // Create jaccard containers to generate into 
  vector<int> dongarra_num_threads = {512};
  gpuErrchk(hipFree( g_d.is ));
  // Create the edge list in the structure needed by algorithm
  vid_t * rowidxJ_h = new vid_t[g.m], *colidxJ_h = new vid_t[g.m];
  dongarra::generate_nonzero_arrays(rowidxJ_h, colidxJ_h, g.xadj, g.adj,g.n);
  // Create the containers of the edge list on GPU
  vid_t * rowidxJ_d, *colidxJ_d;
  gpuErrchk( vcudaMalloc((void**)&rowidxJ_d, sizeof(vid_t) * g.m ) );
  gpuErrchk( vcudaMalloc((void**)&colidxJ_d, sizeof(vid_t) * g.m ) );
  gpuErrchk( hipMemcpy(rowidxJ_d, rowidxJ_h, sizeof(vid_t) * g.m, hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(colidxJ_d, colidxJ_h, sizeof(vid_t) * g.m, hipMemcpyHostToDevice) );
  // calculate jaccards
  for (auto dongarra_threads : dongarra_num_threads){
    total_time = 0;
    for (int i = 0; i< num_average; i++){
      gpuErrchk( hipMemset(emetrics_cuda_d, 0, sizeof(jac_t) * g.m * 1) );
      start = omp_get_wtime();
      int my_no_blocks = g.m/dongarra_threads+(g.m%NUM_THREADS!=0);
      dim3 grid(my_no_blocks, 1,1);
      dongarra::dongarra_jaccard<<<grid,dongarra_threads>>>(g.n, g.n, g.m, rowidxJ_d, colidxJ_d, emetrics_cuda_d, g_d.xadj, g_d.adj, (jac_t*)NULL);
      gpuErrchk( hipMemcpy(emetrics_cuda, emetrics_cuda_d, sizeof(jac_t) * g.m, hipMemcpyDeviceToHost) );
      end = omp_get_wtime();
      total_time+=end-start;
    }
    validate_and_write(g,  "GPU - Dongarra - "+to_string(dongarra_threads)+" threads", emetrics, emetrics_cuda, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);
  }

  gpuErrchk( hipFree(rowidxJ_d) );
  gpuErrchk( hipFree(colidxJ_d) );
  gpuErrchk( vcudaMalloc((void**)&g_d.is, sizeof(vid_t) * g.m ) );
  gpuErrchk( hipMemcpy(g_d.is, g.is, sizeof(vid_t) * g.m, hipMemcpyHostToDevice) );
#endif
#if defined(_CUGRAPH) || defined(_INHOUSE_CUGRAPH)
  //cugraph::GraphCSRView<vid_t, vid_t, jac_t> cuCSR (xadj_d, adj_d, nullptr, g.n, g.m); 
  total_time = 0;
  for (int i = 0; i< num_average; i++){
    gpuErrchk( hipMemset(emetrics_cuda_d, 0, sizeof(jac_t) * g.m * 1) );
    start = omp_get_wtime();
    #ifdef _INHOUSE_CUGRAPH
    inhouse_cugraph::cugraph_jaccard<false, vid_t, vid_t, float>(g_d.is, g_d.xadj, g_d.adj, g.n, g.m, emetrics_cuda_d);
    inhouse_cugraph::cugraph_jaccard_nosum<false, vid_t, vid_t, float>(g_d.is, g_d.xadj, g_d.adj, g.n, g.m, emetrics_cuda_d);
    #else
    cugraph::jaccard(cuCSR, (jac_t*)NULL, emetrics_cuda_d);
    #endif
    //for (int j =0;j <3; j++) total_times[j]+=times[j];
    gpuErrchk( hipMemcpy(emetrics_cuda, emetrics_cuda_d, sizeof(jac_t) * g.m, hipMemcpyDeviceToHost) );
    end = omp_get_wtime();
    total_time+=end-start;
  }
  validate_and_write(g,  "GPU - cuGraph", emetrics, emetrics_cuda, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);
#endif
  
#ifdef _BINNING
  //Each binning experiment will
  cout << "##############################" << endl << "###### Binning #####" << endl;
  gpuErrchk( hipMemset(emetrics_cuda_d, 0, sizeof(jac_t) * g.m * 1) );
  JSONWrapper binning_experiment_json = read_json(binning_experiment_json_file_name);
  vector<tuple<string, vector<tuple<JAC_FUNC<_DIRECTED, vid_t, vid_t, jac_t>, dim3, dim3, vid_t, JSONWrapper>>, SEP_FUNC<vid_t, vid_t>>> all_kernels;
  vector<tuple<JAC_FUNC<_DIRECTED, vid_t, vid_t, jac_t>, dim3, dim3, vid_t, JSONWrapper>> kernels;
  string name;
  vector<vid_t> ranges = binning_experiment_json.Get<std::vector<vid_t>>("ranges");//{32, 64, 128, 256, 512, 1024, 2048, 4096, 8192, 16384, 32768, 65536, 131072, 262144}; 
  dim3 block(1,1,1), grid(1,1,1);
  output_json.SetJSONNested("experiments", "binning", JSONWrapper());
  output_json.NestedSet("experiments", "binning", "ranges", ranges);


  // for each range, add either this kernel or a fallback kernel (in case SM doesn't work etc.
//////////////////////////////////////////////////////////////////////////
    // SMALL
  if (binning_experiment_json.contains("small-sm")){
    vector<int> g_values = binning_experiment_json.NestedGet<vector<int>>("small-sm", "g");
    vector<int> a_values = binning_experiment_json.NestedGet<vector<int>>("small-sm", "a");
    for (auto k : g_values){
      for (auto j : a_values){
        for (int i =0; i< ranges.size(); i++){
            dim3 block(1,1,1), grid(1,1,1); 
            block.x = k; block.y = max(1, WARP_SIZE/block.x); block.z = 1;
            grid.x = max(1, j/block.y);
            int g = block.x, a = block.y*grid.x;
            int sm_fac = ranges[i]; 
            if (sm_fac*sizeof(vid_t) <= max_sm){
              JSONWrapper information = generate_json("u-per-grid-bst-inv-sm", g, a, ranges[i], grid, block, sm_fac);
              kernels.push_back(make_tuple(jac_binning_gpu_u_per_grid_bst_inv_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, sm_fac, information));
            } else {
              dim3 block(1,1,1), grid(1,1,1); 
              block.x = k; block.y = 1; block.z = 1;
              grid.x = j;
              JSONWrapper information = generate_json("u-per-grid-bst-bigsgroup", k, j, ranges[i], grid, block, 1000);
              kernels.push_back(make_tuple(
                    jac_binning_gpu_u_per_grid_bst_bigsgroup_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
            }
        }
        block.x = k; block.y = 1; block.z = 1;
        grid.x = j; 
        JSONWrapper information = generate_json("u-per-grid-bst-bigsgroup", k, j, ranges[ranges.size()-1], grid, block, 1000);
        kernels.push_back(make_tuple(
                                        jac_binning_gpu_u_per_grid_bst_bigsgroup_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
        all_kernels.push_back(make_tuple("small-sm-sg"+string(1,(char)((int)log2(k)+'a'))+to_string(k)+"-sa"+string(1,(char)((int)log2(j)+'a'))+to_string(j),kernels, split_vertices_by_ranges_cugraph_heur<vid_t, vid_t>));
        kernels.clear();
      }
    }
  }
/////////////////////////////////////////////////////////////////////////
  if (binning_experiment_json.contains("small")){
    vector<int> g_values = binning_experiment_json.NestedGet<vector<int>>("small","g");
    vector<int> a_values = binning_experiment_json.NestedGet<vector<int>>("small", "a");
    for (auto k : g_values){
      for (auto j : a_values){
        for (int i =0; i< ranges.size(); i++){
            dim3 block(1,1,1), grid(1,1,1); 
            block.x = k; block.y = max(1, WARP_SIZE/block.x); block.z = 1;
            grid.x = max(1, j/block.y);
            int g = block.x, a = block.y*grid.x;
            JSONWrapper information = generate_json("u-per-grid-bst", g, a, ranges[i], grid, block, 1000);
            kernels.push_back(make_tuple(
                                            jac_binning_gpu_u_per_grid_bst_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
        }
        block.x = k; block.y = 1; block.z = 1;
        grid.x = j; 
        JSONWrapper information = generate_json("u-per-grid-bst", k, j, ranges[ranges.size()-1], grid, block, 1000);
        kernels.push_back(make_tuple(
                                        jac_binning_gpu_u_per_grid_bst_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
        all_kernels.push_back(make_tuple("small-nosm-sg"+string(1, (char)((int)log2(k)+'a'))+to_string(k)+"-sa"+string(1,(char)((int)log2(j)+'a'))+to_string(j),kernels, split_vertices_by_ranges_cugraph_heur<vid_t, vid_t>));
        kernels.clear();
      }
    }
  }
/////////////////////////////////////////////////////////////////////////
// LARGE
  if (binning_experiment_json.contains("large")){
    vector<int> g_values = binning_experiment_json.NestedGet<vector<int>>("large", "g");
    vector<int> a_values = binning_experiment_json.NestedGet<vector<int>>("large", "a");
    for (auto k : g_values){
      for (auto j : a_values){
        for (int i =0; i< ranges.size(); i++){
            dim3 block(1,1,1), grid(1,1,1); 
            block.x = k; block.y = 1; block.z = 1;
            grid.x = j;
            JSONWrapper information = generate_json("u-per-grid-bst-bigsgroup", k, j, ranges[i], grid, block, 1000);
            kernels.push_back(make_tuple( 
                                            jac_binning_gpu_u_per_grid_bst_bigsgroup_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
        }
        block.x = k; block.y = 1; block.z = 1;
        grid.x = j; 
        JSONWrapper information = generate_json("u-per-grid-bst-bigsgroup", k, j, ranges[ranges.size()-1], grid, block, 1000);
        kernels.push_back(make_tuple(
                                        jac_binning_gpu_u_per_grid_bst_bigsgroup_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
        all_kernels.push_back(make_tuple("large-nosm-sg"+string(1, (char)((int)log2(k)+'a'))+to_string(k)+"-sa"+string(1,(char)((int)log2(j)+'a'))+to_string(j),kernels, split_vertices_by_ranges_cugraph_heur<vid_t, vid_t>));
        kernels.clear();
      }
    }
  }
/////////////////////////////////////////////////////////////////////////
  if (binning_experiment_json.contains("large-sm")){
    vector<int> g_values = binning_experiment_json.NestedGet<vector<int>>("large-sm", "g");
    vector<int> a_values = binning_experiment_json.NestedGet<vector<int>>("large-sm","a");
    for (auto k : g_values){
      for (auto j : a_values){
      for (int i =0; i< ranges.size(); i++){
          dim3 block(1,1,1), grid(1,1,1); 
          block.x = k; block.y = 1; block.z = 1;
          grid.x = j;
          int sm_fac = ranges[i]; 
          if (sm_fac*sizeof(vid_t)+block.x/WARP_SIZE*sizeof(vid_t) <= max_sm){
            JSONWrapper information = generate_json("u-per-grid-bst-inv-sm-biggroup", k, j, ranges[i], grid, block, sm_fac);
            kernels.push_back(make_tuple(jac_binning_gpu_u_per_grid_bst_inv_sm_bigsgroup_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, sm_fac, information));
          } else {
            dim3 block(1,1,1), grid(1,1,1); 
            block.x = k; block.y = 1; block.z = 1;
            grid.x = j;
            JSONWrapper information = generate_json("u-per-grid-bst-bigsgroup", k, j, ranges[i], grid, block, 1000);
            kernels.push_back(make_tuple(
                                             jac_binning_gpu_u_per_grid_bst_bigsgroup_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
          }
      }
      block.x = k; block.y = 1; block.z = 1;
      grid.x = j; 
      JSONWrapper information = generate_json("u-per-grid-bst-bigsgroup", k, j, ranges[ranges.size()-1], grid, block, 1000);
      kernels.push_back(make_tuple( 
                                       jac_binning_gpu_u_per_grid_bst_bigsgroup_sm_driver<_DIRECTED, vid_t, vid_t, jac_t>, grid, block, 1000, information));
      all_kernels.push_back(make_tuple("large-sm-sg"+string(1, (char)((int)log2(k)+'a'))+to_string(k)+"-sa"+string(1,(char)((int)log2(j)+'a'))+to_string(j),kernels, split_vertices_by_ranges_cugraph_heur<vid_t, vid_t>));
      kernels.clear();
    }
  }
  }
/////////////////////////////////////////////////////////////////////////
  vector<tuple<pair<unsigned long long, unsigned long long>, double, JSONWrapper>> kernel_time;
  vector<vector<tuple<pair<unsigned long long, unsigned long long>, double, JSONWrapper>>> kernel_times;
  for (auto kernel_splitter : all_kernels){
    total_time = 0;
    auto name = get<0>(kernel_splitter);
    auto one_kernels = get<1>(kernel_splitter);
    auto splitter_function = get<2>(kernel_splitter);
    for (int i =0; i<num_average; i++){
      start = omp_get_wtime();
      kernel_time = binning_based_jaccard<_DIRECTED, vid_t, vid_t, jac_t>(g_d.is, g_d.xadj, g_d.adj, g_d.tadj, g_d.xadj_start, emetrics_cuda_d, g.is, g.xadj, g.adj, g.tadj, g.xadj_start, emetrics_cuda, g.n, g.m, splitter_function, ranges, one_kernels);
      end = omp_get_wtime();
      total_time+=end-start;
      kernel_times.push_back(kernel_time);
    }
    kernel_time = average_kernel_times(kernel_times);
    kernel_times.clear();
    end = total_time/num_average;
    validate_and_write_binning(g,  kernel_time, name, emetrics, emetrics_cuda, total_time, num_average, output_json_file_name, output_json, jaccards_output_path, have_correct);
    kernel_time.clear();
    gpuErrchk( hipMemset(emetrics_cuda_d, 0, sizeof(jac_t) * g.m * (ull)1) );
  }
#endif
#endif
    return 0;
}
